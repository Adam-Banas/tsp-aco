#include "hip/hip_runtime.h"
#include "AcoAlgorithmGpu.hpp"

#include <iostream>

#include "Utils.hpp"

namespace aco {

// TODO: Kernels should go to a separate file and be tested.

// TODO: Elementwise kernels (calculate_edge_scores, evaporate) don't need to be two-dimensional.
// Verify if it would be faster to just make them one-dimensional.

// Kernel that calculates scores for travelling from city to city
__global__ void kernel_calculate_edge_scores(int* costs, float* pheromones, float* out_scores,
                                             std::size_t nodes) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < nodes && y < nodes && x != y) {
        auto i = x * nodes + y;
        out_scores[i] = pheromones[i] / costs[i];
    }
}

// Updates pheromones on all edges due to evaporation
__global__ void kernel_evaporate(float* pheromones, float evaporation_coefficient,
                                 float min_pheromone, std::size_t nodes) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < nodes && y < nodes && x != y) {
        auto i = x * nodes + y;
        auto updated = pheromones[i] * evaporation_coefficient;
        pheromones[i] = updated > min_pheromone ? updated : min_pheromone;
    }
}

// TODO: This could be further parallelized.
// At the moment, each thread calculates pheromones for one agent path. It consists of two parts:
// - calculating the total distance,
// - using this distance to update pheromone on every edge on that path.
// Both above parts can be done in parallel by cities_count threads (the first part may be tricky
// because of synchronization).
__global__ void kernel_add_ants_pheromones(int* costs, float* pheromones, std::size_t* paths,
                                           std::size_t total_threads, std::size_t path_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total_threads) {
        // The path for the current thread
        std::size_t* path = paths + i * path_size;

        // Step 1: Calculate total pheromone left by this agent
        int total_distance = 0;
        for (std::size_t j = 0; j < path_size; ++j) {
            // Path stores visited cities in order. It is a round trip, so the last distance is
            // from the last city directly to the first one.
            auto src = path[j];
            auto dst = path[(j + 1) % path_size];

            total_distance += costs[src * path_size + dst];
        }
        float total_pheromone = 1.f / total_distance;

        // Step 2: Update all visited edges
        for (std::size_t j = 0; j < path_size; ++j) {
            // Path stores visited cities in order. It is a round trip, so the last distance is
            // from the last city directly to the first one.
            auto src = path[j];
            auto dst = path[(j + 1) % path_size];

            // Calculate pheromone to leave on this section and update two-ways
            // Atomic add is needed, because other threads can update the same edges
            int   index_1 = src * path_size + dst;
            int   index_2 = dst * path_size + src;
            float pheromone_to_leave = total_pheromone / costs[index_1];
            atomicAdd(pheromones + index_1, pheromone_to_leave);
            atomicAdd(pheromones + index_2, pheromone_to_leave);
        }
    }
}

// Initialize shortest path just to be valid
static auto make_valid_path(const Graph& graph) {
    Algorithm::Path result(graph.get_size());
    std::iota(begin(result), end(result), 0);
    return result;
}

static void initialize_cuda() {
    auto res = hipSetDevice(0);

    if (res != hipSuccess) {
        std::cerr << "Failed to initialize CUDA! Error code: " << hipGetErrorString(res) << "\n";
        throw std::runtime_error("Failed to initialize CUDA");
    }
}

// Allocate a buffer of given type on the device
template <typename T> static T* allocate_on_device(std::size_t num_elements) {
    auto size_in_bytes = num_elements * sizeof(T);
    T*   buffer = nullptr;
    auto res = hipMalloc((void**)&buffer, size_in_bytes);

    if (res != hipSuccess) {
        std::cerr << "Failed to allocate device buffer! Error code: " << hipGetErrorString(res)
                  << "\n";
        throw std::runtime_error("Failed to allocate device buffer");
    }

    return buffer;
}

template <typename T> void free_device_buffer(T* buffer) {
    auto res = hipFree(buffer);
    if (res != hipSuccess) {
        std::cerr << "Failed to free device buffer! Error code: " << hipGetErrorString(res)
                  << "\n";
    }
}

// Send buffer from host to device
template <typename T> static void send_to_device(T* dst, const std::vector<T>& src) {
    auto size_in_bytes = src.size() * sizeof(T);
    auto res = hipMemcpy(dst, src.data(), size_in_bytes, hipMemcpyHostToDevice);

    if (res != hipSuccess) {
        std::cerr << "Failed to send buffer to device! Error code: " << hipGetErrorString(res)
                  << "\n";
        throw std::runtime_error("Failed to send buffer to device");
    }
}

// Assumes vector of vectors of the same size
template <typename T> static void send_to_device(T* dst, const std::vector<std::vector<T>>& src) {
    if (src.empty()) {
        // Nothing to send
        return;
    }

    auto           chunks = src.size();
    auto           chunk_size = src[0].size();
    auto           elements_count = chunks * chunk_size;
    std::vector<T> intermediate(elements_count);
    for (std::size_t i = 0; i < chunks; ++i) {
        for (std::size_t j = 0; j < chunk_size; ++j) {
            intermediate[i * chunk_size + j] = src[i][j];
        }
    }
    auto buffer_size_in_bytes = intermediate.size() * sizeof(T);
    auto res = hipMemcpy(dst, intermediate.data(), buffer_size_in_bytes, hipMemcpyHostToDevice);

    if (res != hipSuccess) {
        std::cerr << "Failed to send buffer to device! Error code: " << hipGetErrorString(res)
                  << "\n";
        throw std::runtime_error("Failed to send buffer to device");
    }
}

// Send buffer from device to host
template <typename T> static void send_to_host(std::vector<T>& dst, T* src) {
    auto size_in_bytes = dst.size() * sizeof(T);
    auto res = hipMemcpy(dst.data(), src, size_in_bytes, hipMemcpyDeviceToHost);

    if (res != hipSuccess) {
        std::cerr << "Failed to send buffer from device to host! Error code: "
                  << hipGetErrorString(res) << "\n";
        throw std::runtime_error("Failed to send buffer from device to host");
    }
}

AlgorithmGpu::AlgorithmGpu(std::mt19937& random_generator, Graph graph_arg, Config config_arg)
    : Algorithm(random_generator, std::move(graph_arg), config_arg), shortest_path(),
      costs(nullptr), pheromones(nullptr), scores(nullptr), paths(nullptr) {
    shortest_path = make_valid_path(graph);

    // Initialize CUDA, allocate buffers
    initialize_cuda();
    auto nodes = graph.get_size();
    auto edges = nodes * nodes;
    costs = allocate_on_device<int>(edges);
    pheromones = allocate_on_device<float>(edges);
    scores = allocate_on_device<float>(edges);
    paths = allocate_on_device<std::size_t>(config.agents_count * nodes);

    // Send costs from host graph to device (these never change, so it can be done just once)
    send_to_device(costs, graph.costs);
}

AlgorithmGpu::~AlgorithmGpu() {
    // TODO: Some abstraction for buffers would be useful
    free_device_buffer(costs);
    free_device_buffer(pheromones);
    free_device_buffer(scores);
}

const Graph& AlgorithmGpu::get_graph() const {
    // This is not a problem at the moment, but will be soon. Commit that stores pheromones directly
    // on device is coming.
    std::cerr << "[WARN] No synchronization AlgorithmGpu::get_graph\n";
    return graph;
}

const AlgorithmGpu::Path& AlgorithmGpu::get_shortest_path() const {
    return shortest_path;
}

AlgorithmGpu::Path AlgorithmGpu::advance() {
    auto cities = graph.get_size();
    Path iteration_best = make_valid_path(graph);

    // Calculate path scores on GPU.
    // It works slower than CPU counterpart, because there's a lot of data movement.
    auto path_scores = calculate_path_scores();

    // Generate solutions
    std::vector<Path> paths(config.agents_count);
    for (std::size_t i = 0; i < config.agents_count; ++i) {
        auto& path = paths[i];

        // Start from a city with index 'i', modulo in case the number of agents is higher than the
        // number of cities
        path.push_back(i % cities);

        // Choose one new destination in every iteration
        while (path.size() < cities) {
            // Calculate the score (desire to go) for every city
            auto               current_city = path.back();
            std::vector<float> scores(cities);
            for (std::size_t j = 0; j < cities; ++j) {
                if (utils::contains(path, j)) {
                    // Path already visited - leave it a score of zero
                    continue;
                }

                scores[j] = path_scores[current_city * cities + j];
            }

            // Choose the target city using roullette random algorithm
            auto target = utils::roullette(scores, gen);
            path.push_back(target);
        }

        // Path calculated - remember it if is shorter than the current best
        if (path_length(path) < path_length(iteration_best)) {
            iteration_best = path;
        }
    }

    // Update pheromones
    update_pheromones(paths);

    // If the iteration best path is shortest than the global shortest (best so far), remember it
    if (path_length(iteration_best) < path_length(shortest_path)) {
        shortest_path = iteration_best;
    }

    return iteration_best;
}

int AlgorithmGpu::path_length(const Path& path) const {
    int length = 0;
    for (int i = 0; i < path.size(); ++i) {
        // Path stores visited cities in order. It is a round trip, so the last distance is from the
        // last city directly to the first one
        auto src = path[i];
        auto dst = path[(i + 1) % path.size()];

        length += graph.get_cost(src, dst);
    }

    return length;
}

// Calculate on GPU. Works probably much slower than CPU, because of all these allocations and data
// transfers.
std::vector<float> AlgorithmGpu::calculate_path_scores() const {
    auto cities = graph.get_size();
    auto buffer_size = cities * cities;

    // Send data to device
    // TODO: In general, pheromones can be stored and updated directly on device
    send_to_device(pheromones, graph.pheromones);

    // Launch kernel
    auto threads_per_block = 16;
    dim3 block_size(threads_per_block, threads_per_block);
    auto blocks_per_grid_dim =
        (buffer_size + threads_per_block - 1) / threads_per_block; // Rounded up
    dim3 blocks_per_grid(blocks_per_grid_dim, blocks_per_grid_dim);
    kernel_calculate_edge_scores<<<blocks_per_grid, block_size>>>(costs, pheromones, scores,
                                                                  cities);

    auto res = hipGetLastError();
    if (res != hipSuccess) {
        std::cerr << "Failed to launch scores calculation kernel! Error code: "
                  << hipGetErrorString(res) << "\n";
        throw std::runtime_error("Failed to launch scores calculation kernel!");
    }

    // Get data
    std::vector<float> scores_host(buffer_size);
    send_to_host(scores_host, scores);

    return scores_host;
}

void AlgorithmGpu::update_pheromones(const std::vector<Path>& paths) {
    // Step 1: evaporation
    evaporate();

    // Step 2: Pheromones left by ants.
    add_ants_pheromones(paths);
}

void AlgorithmGpu::evaporate() {
    auto cities = graph.get_size();
    auto buffer_size = cities * cities;

    // Send data to device
    // TODO: In general, pheromones can be stored and updated directly on device
    send_to_device(pheromones, graph.pheromones);

    // Launch kernel
    auto threads_per_block = 16;
    dim3 block_size(threads_per_block, threads_per_block);
    auto blocks_per_grid_dim =
        (buffer_size + threads_per_block - 1) / threads_per_block; // Rounded up
    dim3 blocks_per_grid(blocks_per_grid_dim, blocks_per_grid_dim);
    kernel_evaporate<<<blocks_per_grid, block_size>>>(pheromones, config.pheromone_evaporation,
                                                      graph.initial_pheromone, cities);

    auto res = hipGetLastError();
    if (res != hipSuccess) {
        std::cerr << "Failed to launch evaporation kernel! Error code: " << hipGetErrorString(res)
                  << "\n";
        throw std::runtime_error("Failed to launch evaporation kernel!");
    }

    // Send to host
    send_to_host(graph.pheromones, pheromones);
}

void AlgorithmGpu::add_ants_pheromones(const std::vector<Path>& travelled_paths) {
    auto cities = graph.get_size();
    auto total_threads = config.agents_count;

    // Send data to device
    // TODO: In general, pheromones can be stored and updated directly on device
    send_to_device(pheromones, graph.pheromones);
    send_to_device(paths, travelled_paths);

    // Launch kernel
    int  block_size = 256;
    auto blocks_per_grid = (total_threads + block_size - 1) / block_size; // Rounded up
    kernel_add_ants_pheromones<<<blocks_per_grid, block_size>>>(costs, pheromones, paths,
                                                                total_threads, cities);

    auto res = hipGetLastError();
    if (res != hipSuccess) {
        std::cerr << "Failed to launch add_ants_pheromones kernel! Error code: "
                  << hipGetErrorString(res) << "\n";
        throw std::runtime_error("Failed to launch add_ants_pheromones kernel!");
    }

    // Send pheromones back to host
    send_to_host(graph.pheromones, pheromones);
}

} // namespace aco