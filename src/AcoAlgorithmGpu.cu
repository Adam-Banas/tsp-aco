#include "hip/hip_runtime.h"
#include "AcoAlgorithmGpu.hpp"

#include <iostream>

#include "Utils.hpp"

namespace aco {

// Kernel that calculates scores for travelling from city to city
__global__ void calculate_edge_scores(int* costs, float* pheromones, float* out_scores,
                                      std::size_t nodes) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < nodes && y < nodes && x != y) {
        auto i = x * nodes + y;
        out_scores[i] = pheromones[i] / costs[i];
    }
}

// Initialize shortest path just to be valid
static auto make_valid_path(const Graph& graph) {
    Algorithm::Path result(graph.get_size());
    std::iota(begin(result), end(result), 0);
    return result;
}

static void initialize_cuda() {
    auto res = hipSetDevice(0);

    if (res != hipSuccess) {
        std::cerr << "Failed to initialize CUDA! Error code: " << hipGetErrorString(res) << "\n";
        throw std::runtime_error("Failed to initialize CUDA");
    }
}

// Allocate a buffer of given type on the device
template <typename T> static T* allocate_on_device(std::size_t num_elements) {
    auto size_in_bytes = num_elements * sizeof(T);
    T*   buffer = nullptr;
    auto res = hipMalloc((void**)&buffer, size_in_bytes);

    if (res != hipSuccess) {
        std::cerr << "Failed to allocate device buffer! Error code: " << hipGetErrorString(res)
                  << "\n";
        throw std::runtime_error("Failed to allocate device buffer");
    }

    return buffer;
}

template <typename T> void free_device_buffer(T* buffer) {
    auto res = hipFree(buffer);
    if (res != hipSuccess) {
        std::cerr << "Failed to free device buffer! Error code: " << hipGetErrorString(res)
                  << "\n";
    }
}

// Send buffer from host to device
template <typename T> static void send_to_device(T* dst, const std::vector<T>& src) {
    auto size_in_bytes = src.size() * sizeof(T);
    auto res = hipMemcpy(dst, src.data(), size_in_bytes, hipMemcpyHostToDevice);

    if (res != hipSuccess) {
        std::cerr << "Failed to send buffer to device! Error code: " << hipGetErrorString(res)
                  << "\n";
        throw std::runtime_error("Failed to send buffer to device");
    }
}

// Send buffer from device to host
template <typename T> static void send_to_host(std::vector<T>& dst, T* src) {
    auto size_in_bytes = dst.size() * sizeof(T);
    auto res = hipMemcpy(dst.data(), src, size_in_bytes, hipMemcpyDeviceToHost);

    if (res != hipSuccess) {
        std::cerr << "Failed to send buffer from device to host! Error code: "
                  << hipGetErrorString(res) << "\n";
        throw std::runtime_error("Failed to send buffer from device to host");
    }
}

AlgorithmGpu::AlgorithmGpu(std::mt19937& random_generator, Graph graph_arg, Config config_arg)
    : Algorithm(random_generator, std::move(graph_arg), config_arg), shortest_path(),
      costs(nullptr), pheromones(nullptr), scores(nullptr) {
    shortest_path = make_valid_path(graph);

    // Initialize CUDA, allocate buffers
    initialize_cuda();
    auto buffer_size = graph.get_size() * graph.get_size();
    costs = allocate_on_device<int>(buffer_size);
    pheromones = allocate_on_device<float>(buffer_size);
    scores = allocate_on_device<float>(buffer_size);

    // Send costs from host graph to device (these never change, so it can be done just once)
    send_to_device(costs, graph.costs);
}

AlgorithmGpu::~AlgorithmGpu() {
    // TODO: Some abstraction for buffers would be useful
    free_device_buffer(costs);
    free_device_buffer(pheromones);
    free_device_buffer(scores);
}

const Graph& AlgorithmGpu::get_graph() const {
    return graph;
}

const AlgorithmGpu::Path& AlgorithmGpu::get_shortest_path() const {
    return shortest_path;
}

AlgorithmGpu::Path AlgorithmGpu::advance() {
    auto cities = graph.get_size();
    Path iteration_best = make_valid_path(graph);

    // Calculate path scores on GPU.
    // It works slower than CPU counterpart, because there's a lot of data movement.
    auto path_scores = calculate_path_scores();

    // Generate solutions
    std::vector<Path> paths(config.agents_count);
    for (std::size_t i = 0; i < config.agents_count; ++i) {
        auto& path = paths[i];

        // Start from a city with index 'i', modulo in case the number of agents is higher than the
        // number of cities
        path.push_back(i % cities);

        // Choose one new destination in every iteration
        while (path.size() < cities) {
            // Calculate the score (desire to go) for every city
            auto               current_city = path.back();
            std::vector<float> scores(cities);
            for (std::size_t j = 0; j < cities; ++j) {
                if (utils::contains(path, j)) {
                    // Path already visited - leave it a score of zero
                    continue;
                }

                scores[j] = path_scores[current_city * cities + j];
            }

            // Choose the target city using roullette random algorithm
            auto target = utils::roullette(scores, gen);
            path.push_back(target);
        }

        // Path calculated - remember it if is shorter than the current best
        if (path_length(path) < path_length(iteration_best)) {
            iteration_best = path;
        }
    }

    // Update pheromones
    // Step 1: evaporation
    graph.update_all(config.pheromone_evaporation);

    // Step 2: Pheromones left by ants.
    // Basic algorithm, where every ant leaves pheromones, and the amount is independent from
    // other ants' solutions.
    // No limit on total pheromone on a section.
    for (const auto& path : paths) {
        // The total amount of pheromone left by ant is inversely proportional to the distance
        // covered by ant.
        float total_pheromone = 1.f / path_length(path);

        for (int i = 0; i < path.size(); ++i) {
            // Path stores visited cities in order. It is a round trip, so the last distance is
            // from the last city directly to the first one
            auto src = path[i];
            auto dst = path[(i + 1) % path.size()];

            // The amount of pheromone to leave is proportional to the section length
            float pheromone_to_leave = total_pheromone / graph.get_cost(src, dst);
            graph.add_pheromone_two_way(src, dst, pheromone_to_leave);
        }
    }

    // If the iteration best path is shortest than the global shortest (best so far), remember it
    if (path_length(iteration_best) < path_length(shortest_path)) {
        shortest_path = iteration_best;
    }

    return iteration_best;
}

int AlgorithmGpu::path_length(const Path& path) const {
    int length = 0;
    for (int i = 0; i < path.size(); ++i) {
        // Path stores visited cities in order. It is a round trip, so the last distance is from the
        // last city directly to the first one
        auto src = path[i];
        auto dst = path[(i + 1) % path.size()];

        length += graph.get_cost(src, dst);
    }

    return length;
}

// Calculate on GPU. Works probably much slower than CPU, because of all these allocations and data
// transfers.
std::vector<float> AlgorithmGpu::calculate_path_scores() const {
    auto cities = graph.get_size();
    auto buffer_size = cities * cities;

    // Send data to device
    // TODO: In general, pheromones can be stored and updated directly on device
    send_to_device(pheromones, graph.pheromones);

    // Launch kernel
    auto threads_per_block = 16;
    dim3 block_size(threads_per_block, threads_per_block);
    auto blocks_per_grid_dim =
        (buffer_size + threads_per_block + 1) / threads_per_block; // Rounded up
    dim3 blocks_per_grid(blocks_per_grid_dim, blocks_per_grid_dim);
    calculate_edge_scores<<<blocks_per_grid, block_size>>>(costs, pheromones, scores, cities);

    auto res = hipGetLastError();
    if (res != hipSuccess) {
        std::cerr << "Failed to launch multiplication kernel! Error code: "
                  << hipGetErrorString(res) << "\n";
        throw std::runtime_error("Failed to launch multiplication kernel");
    }

    // Get data
    std::vector<float> scores_host(buffer_size);
    send_to_host(scores_host, scores);

    return scores_host;
}

} // namespace aco