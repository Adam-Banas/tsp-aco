#include "hip/hip_runtime.h"
#include "AcoAlgorithmGpu.hpp"

#include <iostream>

#include "Utils.hpp"

namespace aco {

// Initialize shortest path just to be valid
static auto make_valid_path(const Graph& graph) {
    Algorithm::Path result(graph.get_size());
    std::iota(begin(result), end(result), 0);
    return result;
}

AlgorithmGpu::AlgorithmGpu(std::mt19937& random_generator, Graph graph_arg, Config config_arg)
    : Algorithm(random_generator, std::move(graph_arg), config_arg), shortest_path() {
    shortest_path = make_valid_path(graph);
}

const Graph& AlgorithmGpu::get_graph() const {
    return graph;
}

const AlgorithmGpu::Path& AlgorithmGpu::get_shortest_path() const {
    return shortest_path;
}

AlgorithmGpu::Path AlgorithmGpu::advance() {
    auto cities = graph.get_size();
    Path iteration_best = make_valid_path(graph);

    // Generate solutions
    std::vector<Path> paths(config.agents_count);
    for (std::size_t i = 0; i < config.agents_count; ++i) {
        auto& path = paths[i];

        // Start from a city with index 'i', modulo in case the number of agents is higher than the
        // number of cities
        path.push_back(i % cities);

        // Choose one new destination in every iteration
        while (path.size() < cities) {
            // Calculate the score (desire to go) for every city
            std::vector<float> path_scores(cities);
            auto               current_city = path.back();
            for (std::size_t j = 0; j < cities; ++j) {
                if (utils::contains(path, j)) {
                    // Path already visited - leave it a score of zero
                    continue;
                }

                // Basic score function without alpha and beta coefficients
                // Basic heuristic - just a reciprocal of the distance, so that shorter paths
                // are preferred in general
                // TODO: Precompute reciprocals of distances?
                path_scores[j] =
                    graph.get_pheromone(current_city, j) / graph.get_cost(current_city, j);
            }

            // Choose the target city using roullette random algorithm
            auto target = utils::roullette(path_scores, gen);
            path.push_back(target);
        }

        // Path calculated - remember it if is shorter than the current best
        if (path_length(path) < path_length(iteration_best)) {
            iteration_best = path;
        }
    }

    // Update pheromones
    // Step 1: evaporation
    graph.update_all(config.pheromone_evaporation);

    // Step 2: Pheromones left by ants.
    // Basic algorithm, where every ant leaves pheromones, and the amount is independent from
    // other ants' solutions.
    // No limit on total pheromone on a section.
    for (const auto& path : paths) {
        // The total amount of pheromone left by ant is inversely proportional to the distance
        // covered by ant.
        float total_pheromone = 1.f / path_length(path);

        for (int i = 0; i < path.size(); ++i) {
            // Path stores visited cities in order. It is a round trip, so the last distance is
            // from the last city directly to the first one
            auto src = path[i];
            auto dst = path[(i + 1) % path.size()];

            // The amount of pheromone to leave is proportional to the section length
            float pheromone_to_leave = total_pheromone / graph.get_cost(src, dst);
            graph.add_pheromone_two_way(src, dst, pheromone_to_leave);
        }
    }

    // If the iteration best path is shortest than the global shortest (best so far), remember it
    if (path_length(iteration_best) < path_length(shortest_path)) {
        shortest_path = iteration_best;
    }

    return iteration_best;
}

int AlgorithmGpu::path_length(const Path& path) const {
    int length = 0;
    for (int i = 0; i < path.size(); ++i) {
        // Path stores visited cities in order. It is a round trip, so the last distance is from the
        // last city directly to the first one
        auto src = path[i];
        auto dst = path[(i + 1) % path.size()];

        length += graph.get_cost(src, dst);
    }

    return length;
}

} // namespace aco